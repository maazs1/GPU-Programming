#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
		    }                                                                     \
      } while (0)

#define Mask_width 5
#define Mask_radius Mask_width / 2
#define clamp(x) (min(max((x), 0.0), 1.0))

__global__ void convolution(float *I, const float *M,
	float *P, int channels, int width, int height) {

	int Col = blockIdx.x * blockDim.x + threadIdx.x;
	int Row = blockIdx.y * blockDim.y + threadIdx.y;
	int radius = Mask_width / 2;
	int maskWidth = 5;

	if (Col < width && Row < height){
		for (int x = 0; x < channels; x++){  //apply mask and image channel for each pixel 
		
			float pixval = 0;
			int xoffset = Row - radius;  //x position
			int yoffset = Col - radius;  //y position

			for (int z = 0; z < maskWidth; z++){   // These for loops will iterate through the mask

				for (int y = 0; y < maskWidth; y++){
					int pixRow = xoffset + z;
					int pixCol = yoffset + y;

					if (pixRow >= 0 && pixRow < height && pixCol >= 0 && pixCol < width){  //will go into if statement if the pixel for mask in inside the image boundary 

						float newVal = I[(pixRow*width + pixCol)*channels + x] * M[(z *radius + y)];
						pixval += newVal;

					}

				}
			}
			P[(Row*width + Col) * channels + x] = clamp(pixval);
		}
	}


}



int main(int argc, char *argv[]) {
	wbArg_t arg;
	int maskRows;
	int maskColumns;
	int imageChannels;
	int imageWidth;
	int imageHeight;
	char *inputImageFile;
	char *inputMaskFile;
	wbImage_t inputImage;
	wbImage_t outputImage;
	float *hostInputImageData;
	float *hostOutputImageData;
	float *hostMaskData;
	float *deviceInputImageData;
	float *deviceOutputImageData;
	float *deviceMaskData;

	arg = wbArg_read(argc, argv); /* parse the input arguments */

	inputImageFile = wbArg_getInputFile(arg, 0);
	inputMaskFile = wbArg_getInputFile(arg, 1);

	inputImage = wbImport(inputImageFile);
	hostMaskData = (float *)wbImport(inputMaskFile, &maskRows, &maskColumns);

	assert(maskRows == 5);    /* mask height is fixed to 5 in this mp */
	assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

	imageWidth = wbImage_getWidth(inputImage);
	imageHeight = wbImage_getHeight(inputImage);
	imageChannels = wbImage_getChannels(inputImage);

	outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

	hostInputImageData = wbImage_getData(inputImage);
	hostOutputImageData = wbImage_getData(outputImage);

	wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

	wbTime_start(GPU, "Doing GPU memory allocation");
	//TODO: INSERT CODE HERE
	hipMalloc((void**)&deviceInputImageData, imageHeight * imageWidth * imageChannels * sizeof(float));
	hipMalloc((void**)&deviceOutputImageData, imageHeight * imageWidth * imageChannels * sizeof(float));
	hipMalloc((void**)&deviceMaskData, maskColumns * maskRows * sizeof(float));
	wbTime_stop(GPU, "Doing GPU memory allocation");

	wbTime_start(Copy, "Copying data to the GPU");
	//TODO: INSERT CODE HERE
	hipMemcpy(deviceInputImageData, hostInputImageData, imageHeight * imageWidth * imageChannels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(deviceMaskData, hostMaskData, maskColumns * maskRows * sizeof(float), hipMemcpyHostToDevice);

	wbTime_stop(Copy, "Copying data to the GPU");

	wbTime_start(Compute, "Doing the computation on the GPU");
	//TODO: INSERT CODE HERE
	dim3 gridSize(ceil((float)imageWidth / 16), ceil((float)imageHeight / 16));
	dim3 blockSize(16,16);
	convolution<< < gridSize, blockSize >> > (deviceInputImageData, deviceMaskData, deviceOutputImageData, imageChannels, imageWidth, imageHeight);

	
	hipDeviceSynchronize();
	wbTime_stop(Compute, "Doing the computation on the GPU");

	wbTime_start(Copy, "Copying data from the GPU");
	//TODO: INSERT CODE HERE
	hipMemcpy(hostOutputImageData, deviceOutputImageData, imageWidth*imageHeight*imageChannels *sizeof(float), hipMemcpyDeviceToHost);
	
	
	wbTime_stop(Copy, "Copying data from the GPU");

	wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

	wbSolution(arg, outputImage);

	//TODO: RELEASE CUDA MEMORY
	hipFree(deviceMaskData);
	hipFree(deviceOutputImageData);
	hipFree(deviceInputImageData);
	
	free(hostMaskData);
	wbImage_delete(outputImage);
	wbImage_delete(inputImage);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}
