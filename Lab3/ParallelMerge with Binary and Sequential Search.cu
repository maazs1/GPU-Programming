#include "hip/hip_runtime.h"
#include <wb.h>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__device__ int binarySearch(const int value, const int *A, const int N)
{
	// TODO: Implement a binary search that returns
	// the index where all values in A are less than
	// the given value.


	int min = 0;
	int max = N;
	int sum = 0;

	if (value > A[N - 1]){
		return N;
	}
	else{
		while (min <= max){

			int guess = min + (max - min) / 2;



			if (value <= A[guess]){
				max = guess - 1;
			}
			
			else if (A[guess]<value){
				sum = guess + 1;
				min = guess + 1;
			}
				
		}

		return sum;
	}
	

}

__device__ int linearSearch(const int value, const int *A, const int N)
{
	int val = 0;
	int index = 0;

	while (index < N){
		if (A[index] < value){
			val+=1;
			index+=1;
		}
		else{
			index+=1;
		}
	}
	return val;
}

__global__ void merge(int *C, const int *A, const int *B, const int N)
{
	// TODO: Merge arrays A and B into C. To make it
	// easier you can assume the following:
	// 
	// 1) A and B are both size N
	//
	// 2) C is size 2N
	//
	// 3) Both A and B are sorted arrays
	//
	// The algorithm should work as follows:
	// Given inputs A and B as follows:
	// A = [0 2 4 10]
	// B = [1 5 7 9]
	//
	// Step 1:
	// Find for each element in array A the index i that
	// would A[i] be inserted in array B or in other 
	// words find the smallest j where A[i] < B[j].

	int i, iA, iB;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N){
		iA = linearSearch(A[i], B, N);
		iB = linearSearch(B[i] + 1, A, N);
		C[i + iA] = A[i];
		C[i + iB] = B[i];
	}

	//
	// Step 2:
	// Do the same for B, but this time find the j 
	// where B[i] < A[j].
	//
	// Step 3:
	// Since we know how many elements come before
	// A[i] in array A and we know how many elements 
	// come before A[i] in array B, which is given by
	// are calculation of j. We should know where A[i]
	// is inserted into C, given i and j.
	//
	// This same logic can be used to find where B[i]
	// should be inserted into C. Although you will have
	// to make a minor change to handle duplicates in A 
	// and B. Or in other words if A and B intersect at 
	// all some values in C will be incorrect. This 
	// occurs because A and B will want to put the values 
	// in the same place in C.
	
	
}

int main(int argc, char **argv) {
	wbArg_t args;
	int N;
	int* A;
	int* B;
	int* C;
	int* deviceA;
	int* deviceB;
	int* deviceC;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	A = (int *)wbImport(wbArg_getInputFile(args, 0), &N, NULL, "Integer");
	B = (int *)wbImport(wbArg_getInputFile(args, 1), &N, NULL, "Integer");
	C = (int *)malloc(2 * N * sizeof(int));
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The input length is ", N);

	int threads = 256;
	int blocks = N / threads + ((N%threads == 0) ? 0 : 1);

	wbTime_start(GPU, "Allocating GPU memory.");
	hipMalloc((void **)&deviceA, N * sizeof(int));
	hipMalloc((void **)&deviceB, N * sizeof(int));
	hipMalloc((void **)&deviceC, 2 * N * sizeof(int));
	wbTime_stop(GPU, "Allocating GPU memory.");


	wbTime_start(GPU, "Copying input memory to the GPU.");
	hipMemcpy(deviceA, A, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceB, B, N * sizeof(int), hipMemcpyHostToDevice);
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	// Perform on CUDA.
	const dim3 blockSize(threads, 1, 1);
	const dim3 gridSize(blocks, 1, 1);

	wbTime_start(Compute, "Performing CUDA computation");
	merge << < gridSize, blockSize >> >(deviceC, deviceA, deviceB, N);
	hipDeviceSynchronize();
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	hipMemcpy(C, deviceC, 2 * N * sizeof(int), hipMemcpyDeviceToHost);
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);
	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, C, 2*N);

	free(A);
	free(B);
	free(C);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}
