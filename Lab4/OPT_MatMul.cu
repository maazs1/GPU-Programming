#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <wb.h>


#define TILE_WIDTH 16

// Compute C = A * B
__global__ void matrixMultiplyShared(
	float *A, float *B, float *C, 
	int numARows, int numAColumns, 
	int numBRows, int numBColumns, 
	int numCRows, int numCColumns) 
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int blockX = blockIdx.x, blockY = blockIdx.y;
	int tileX = threadIdx.x, tileY = threadIdx.y;
	

	__shared__ float As[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

	float value = 0;
	int size =((numAColumns - 1) / TILE_WIDTH + 1);

	for (int a = 0; a< size; ++a){


		if (Row < numARows && a * TILE_WIDTH + tileX < numAColumns){
			As[tileY][tileX] = A[Row * numAColumns +a *TILE_WIDTH  + tileX];
		}
		else{
			As[tileY][tileX] = 0.0;
		}

		if (a * TILE_WIDTH + tileY < numBRows && Col < numBColumns){
			Bs[tileY][tileX] = B[(a*TILE_WIDTH + tileY) * numBColumns + Col];
		}
		else{
			Bs[tileY][tileX] = 0.0;
		}
		
		__syncthreads();

		for (int k = 0; k < TILE_WIDTH; ++k){
			value += As[tileY][k] * Bs[k][tileX];

		}
		__syncthreads();

		
		if (Row < numCRows && Col < numCColumns){
			C[Row* numCColumns + Col] = value;

		}



	}


	// TODO: Insert code to implement matrix multiplication 
	// here you have to use shared memory for this lab.
	// Take a the tiled matrix multiplication. Also we 
	// will be testing the speed up between a basic
	// matrix multiplication and this kernel. To pass 
	// the tests for the tiled matrix multiplication
	// you will need to have the correct output and
	// have a significant speed up over a basic matrix
	// multiplication.
	//
	// HINT: Take a look at the slides
	// HINT: Look at TILE_WIDTH defined above
	
}

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
	    }                                                                     \
    } while (0)

int main(int argc, char **argv) {
	wbArg_t args;
	float *hostA; // The A matrix
	float *hostB; // The B matrix
	float *hostC; // The output C matrix
	float *deviceA;
	float *deviceB;
	float *deviceC;
	int numARows;    // number of rows in the matrix A
	int numAColumns; // number of columns in the matrix A
	int numBRows;    // number of rows in the matrix B
	int numBColumns; // number of columns in the matrix B
	int numCRows;
	int numCColumns;

	args = wbArg_read(argc, argv);

#if LAB_DEBUG
	std::cout << "Running Tiled Matrix Multiplicaion ..." << std::endl;
#endif

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
		&numAColumns);
	hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
		&numBColumns);
	// TODO: Allocate the hostC matrix
	hostC = (float *)malloc((numBColumns*numARows)*sizeof(float));
	
	wbTime_stop(Generic, "Importing data and creating memory on host");

	// TODO: Set numCRows and numCColumns
	numCRows = numARows;
	numCColumns = numBColumns;

	wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
	wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
	wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);

	wbTime_start(GPU, "Allocating GPU memory.");
	// TODO: Allocate GPU memory here
	hipMalloc((void**)&deviceA, (numAColumns*numARows)*sizeof(float));
	hipMalloc((void**)&deviceB, (numBColumns*numBRows)*sizeof(float));
	hipMalloc((void**)&deviceC, (numCColumns*numCRows)*sizeof(float));

	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	// TODO: Copy memory to the GPU here
	hipMemcpy(deviceA, hostA, (numAColumns*numARows)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(deviceB, hostB, (numBColumns*numBRows)*sizeof(float), hipMemcpyHostToDevice);



	wbTime_stop(GPU, "Copying input memory to the GPU.");

	// TODO: Initialize the grid and block dimensions here
	// Here you will have to use dim3
	//
	// HINT: Take a look at the slides
	// HINT: Look at TILE_WIDTH defined at the top
	//
	// dim3 blockDim( ... );
	// dim3 gridDim( ... );

	dim3 block_size(TILE_WIDTH, TILE_WIDTH, 1);

	dim3 grid_size(numCColumns  / TILE_WIDTH + 1, numCRows  / TILE_WIDTH + 1, 1);



	// wbLog(TRACE, "The block dimensions are ", blockDim.x, " x ", blockDim.y);
	// wbLog(TRACE, "The grid dimensions are ", gridDim.x, " x ", gridDim.y);

	wbTime_start(Compute, "Performing CUDA computation");
	// TODO:: Launch the GPU Kernel here
	matrixMultiplyShared << <grid_size, block_size >> > (deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);


	hipDeviceSynchronize();
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	// TODO:: Copy the GPU memory back to the CPU here
	hipMemcpy(hostC, deviceC, (numCColumns*numCRows)*sizeof(float), hipMemcpyDeviceToHost);

	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	// TODO:: Free the GPU memory here
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);

	
	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, hostC, numCRows, numCColumns);

	free(hostA);
	free(hostB);
	free(hostC);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}
